#include "kernels.h"

#define BLOCKSIZE 32

void copyMatrixFromDeviceToHost(void **deviceMatrix, void **matrix, int rows, int cols, MatrixType type)
{
    switch (type)
    {
    case INT:
        for (int i = 0; i < rows; i++)
        {
            hipMemcpy(matrix[i], deviceMatrix[i], cols * sizeof(int), hipMemcpyDeviceToHost);
        }
        break;

    case FLOAT:
        for (int i = 0; i < rows; i++)
        {
            hipMemcpy(matrix[i], deviceMatrix[i], cols * sizeof(float), hipMemcpyDeviceToHost);
        }
        break;

    case DOUBLE:
        for (int i = 0; i < rows; i++)
        {
            hipMemcpy(matrix[i], deviceMatrix[i], cols * sizeof(double), hipMemcpyDeviceToHost);
        }
        break;
    }
}

int main(int argc, char **argv)
{
    int rows, cols;
    MatrixType type;

    // read the matrix from input file
    void **matrix = readMatrix(argv[1], &rows, &cols, &type);

    if (matrix == NULL)
        return 1;

    // set grid and bloxk size
    dim3 block(BLOCKSIZE, BLOCKSIZE);
    dim3 grid((rows + block.x - 1) / block.x);

    void **deviceMatrix;

    switch (type)
    {
    case INT:
        hipMallocManaged(&deviceMatrix, rows * sizeof(int *));
        for (int i = 0; i < rows; i++)
        {
            hipMallocManaged(&(deviceMatrix[i]), cols * sizeof(int));
            hipMemcpy(deviceMatrix[i], matrix[i], cols * sizeof(int), hipMemcpyHostToDevice);
        }
        break;

    case FLOAT:
        hipMallocManaged(&deviceMatrix, rows * sizeof(float *));
        for (int i = 0; i < rows; i++)
        {
            hipMallocManaged(&(deviceMatrix[i]), cols * sizeof(float));
            hipMemcpy(deviceMatrix[i], matrix[i], cols * sizeof(float), hipMemcpyHostToDevice);
        }
        break;

    case DOUBLE:
        hipMallocManaged(&deviceMatrix, rows * sizeof(double *));
        for (int i = 0; i < rows; i++)
        {
            hipMallocManaged(&(deviceMatrix[i]), cols * sizeof(double));
            hipMemcpy(deviceMatrix[i], matrix[i], cols * sizeof(double), hipMemcpyHostToDevice);
        }
        break;
    }
    void* output;
    hipMallocManaged(&output, rows * cols * sizeof(int *));
    

    // launchSortKernel(deviceMatrix, rows, cols, type, block, grid);
    switch (type)
    {
    case INT:
        sortRowsKernelInt<<<grid, block>>>((int **)(deviceMatrix), rows, cols);
        transposeKernelInt<<<grid, block>>>((int**)(deviceMatrix), (int*)output, rows, cols);
        break;
    case FLOAT:
        sortRowsKernelFloat<<<grid, block>>>((float **)(deviceMatrix), rows, cols);
        break;
    case DOUBLE:
        sortRowsKernelDouble<<<grid, block>>>((double **)(deviceMatrix), rows, cols);
        break;
    }


    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            printf("%d ", ((int *)deviceMatrix[i])[j]);
        }
        printf("\n");
    }

    printf("\n");



    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            printf("%d ", ((int *)output)[i * rows + j]);
        }
        printf("\n");
    }

    copyMatrixFromDeviceToHost(deviceMatrix, matrix, rows, cols, type);

    for (int i = 0; i < rows; i++)
    {
        hipFree(deviceMatrix[i]);
    }
    hipFree(deviceMatrix);

    writeMatrix(argv[2], matrix, rows, cols, type);

    return 0;
}

